#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>


#define N 2000000
#define MAX_ERR 1e-6

__global__ void vector_multiply(float *out, float *a, float *b, int n) {
    //TODO
	int i;
	for(i = 0; i<n; i++){
	out[i] = a[i] * b[i];
	}
}

int main(){
    float *a, *b, *out;
    float *d_a, *d_b, *d_out; 

    a   = (float*)malloc(sizeof(float) * N);
    b   = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);

    // Initialize host arrays
    for(int i = 0; i < N; i++){
        a[i] = 4.0f;
        b[i] = 9.0f;
    }

   
    hipMalloc((void**)&d_a, sizeof(float) * N);
    hipMalloc((void**)&d_b, sizeof(float) * N);
    hipMalloc((void**)&d_out, sizeof(float) * N);

    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);

    // Executing kernel 
    //TODO: call vector_multiply<<<1,1>>> with output array (d_out), input arrays (d_a, d_b), and N
   
	vector_multiply<<<1,1>>>(d_out,d_a,d_b,N);
    
   
    hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);

    // Verification
    for(int i = 0; i < N; i++){
        assert(fabs(out[i] - a[i] * b[i]) < MAX_ERR);
    }
    printf("out[1999999] = %f\n", out[1999999]);
    printf("PASSED\n");

    // Deallocate device memory using cudaFree
    //TODO
hipFree(d_a);
hipFree(d_b);
hipFree(d_out);

    // Deallocate host memory using free
   //TODO
free(a);
free(b);
free(out);
}
